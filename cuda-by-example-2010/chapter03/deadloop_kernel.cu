#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"

__global__ void kernel( void ) 
{
	int temp = 0;
	while(1)
		temp++;
}

int main( void ) 
{
	printf( "Dead loop start.\n" );

	kernel<<<1,1>>>();

	printf( "Dead loop end.\n" );
    return 0;
}

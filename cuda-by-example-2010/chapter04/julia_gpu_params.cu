#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

// [2023-09-02] Chj: User need to  assign parameters <dim> and <scale> .

#include "../common/book.h"
#include "../common/cpu_bitmap.h"
#include "../common/chjdbg.h"

int dim = 1000;
float scale = 1.5;

struct hipComplex 
{
    float   r;
    float   i;
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int julia( int x, int y, int dim, float scale ) 
{
    float jx = scale * (float)(dim/2 - x)/(dim/2);
    float jy = scale * (float)(dim/2 - y)/(dim/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel( unsigned char *ptr, int dim, float scale ) 
{
    // map from blockIdx to pixel position
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // now calculate the value at that position
    int juliaValue = julia( x, y, dim, scale );
    ptr[offset*4 + 0] = 255 * juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
};

int main( int argc, char *argv[] ) 
{
	if(argc<3) {
		printf("Need two parameters. \n");
		printf("    julia_gpu_params <sample_points> <scale>\n");
		printf("\n");
		printf("For example:\n");
		printf("    julia_gpu_params 1000 1.5\n");
		printf("    julia_gpu_params 500 5.0\n");
		return 1;
	}

	dim = (int)strtoul(argv[1], nullptr, 0);
	scale = (float)atof(argv[2]);

	if(dim<=0) {
		printf("ERROR: <sample_points> must > 0, given: %d\n", dim);
		return 4;
	}

	if(scale<=0) {
		printf("ERROR: <scale> must > 0, given: %g\n", scale);
		return 4;
	}

	printf("Using sample_points=%d , scale=%g\n", dim, scale);

    DataBlock   data;
    CPUBitmap bitmap( dim, dim, &data );
    unsigned char    *dev_bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap, bitmap.image_size() ) );
    data.dev_bitmap = dev_bitmap;

	unsigned int64 usec_start = ps_GetOsMicrosecs64(); // chj

    dim3    grid(dim, dim);
    kernel<<<grid,1>>>( dev_bitmap, dim, scale );
	
	unsigned int64 usec_done1 = ps_GetOsMicrosecs64(); // chj

    HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), dev_bitmap,
                              bitmap.image_size(),
                              hipMemcpyDeviceToHost ) );

	unsigned int64 usec_done2 = ps_GetOsMicrosecs64(); // chj

	printf("Julia calculation time cost milliseconds (GPU): %s\n", 
		us_to_msecstring(usec_done1 - usec_start));
	printf("hipMemcpyDeviceToHost %d bytes, cost milliseconds: %s\n", 
		bitmap.image_size(), us_to_msecstring(usec_done2 - usec_done1));

    HANDLE_ERROR( hipFree( dev_bitmap ) );
                              
    bitmap.display_and_exit();
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../../share/share.h"

#define BIN256 256
//#define THREADS256 256

__global__ void myhistogram_01( // @page 98
	const unsigned char * d_hist_data,
	unsigned int * d_bin_data,
	int sample_count) 
{
	/* Work out our thread id */
	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	const unsigned int tid = idx + idy * blockDim.x * gridDim.x;

	if(tid<sample_count)
	{
		/* Fetch the data value */
		const unsigned char value = d_hist_data[tid];

//		printf("[#%d] .%u\n", tid, value);

		atomicAdd(&(d_bin_data[value]), 1);
	}
}

__global__ void myhistogram_02( // @page 99-100
	const unsigned int * d_hist_data, // note: each call will process 4 bytes(Uint)
	unsigned int * d_bin_data,
	int sample_ints) 
{
	/* Work out our thread id */
	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	const unsigned int tid = idx + idy * blockDim.x * gridDim.x;

	if(tid<sample_ints)
	{
		/* Fetch the data value */
		const Uint value_u32 = d_hist_data[tid];

		atomicAdd( &(d_bin_data[ (value_u32 & 0x000000FF) ]), 1 );
		atomicAdd( &(d_bin_data[ (value_u32 & 0x0000FF00) >>  8 ]), 1 );
		atomicAdd( &(d_bin_data[ (value_u32 & 0x00FF0000) >> 16 ]), 1 );
		atomicAdd( &(d_bin_data[ (value_u32 & 0xFF000000) >> 24 ]), 1 );
	}
}

__shared__ unsigned int d_bin_data_shared[BIN256];

__global__ void myhistogram_03a( // @page 101 modified
	const unsigned int * d_hist_data,
	unsigned int * const d_bin_data,
	int sample_ints)
{
	// Chj: Note: this program implies threadIdx.y==1
	// Each call copes with four user samples(each sample is one byte).

	/* Work out our thread id */
	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	Uint tid = idx + idy * blockDim.x * gridDim.x;
	Uint threads_per_block = blockDim.x * blockDim.y;
	
	// Chj: Clear the d_bin_data_shared[] array.
	int idxBin = threadIdx.x;
	while(idxBin < BIN256)
	{
		d_bin_data_shared[idxBin] = 0;
		idxBin += threads_per_block;
	}

	// All threads should wait for the above clearing done.
	__syncthreads();

	/* Fetch the data value as 32 bit */
	const unsigned int value_u32 = d_hist_data[tid];
	
	// Partial counting into d_bin_data_shared[]
	//
	if(tid < sample_ints)
	{
		atomicAdd( &(d_bin_data_shared[ (value_u32 & 0x000000FF) ]), 1 );
		atomicAdd( &(d_bin_data_shared[ (value_u32 & 0x0000FF00) >>  8 ]), 1 );
		atomicAdd( &(d_bin_data_shared[ (value_u32 & 0x00FF0000) >> 16 ]), 1 );
		atomicAdd( &(d_bin_data_shared[ (value_u32 & 0xFF000000) >> 24 ]), 1 );
	}
	
	/* Wait for all threads to update shared memory, again */
	__syncthreads();

#if 0
	// Chj: Let the first thread accumulate the counting result. 
	// ( This slows down the overall myhistogram_03a() by 5~10X. Very bad.)
	if(threadIdx.x==0)
	{
		for(int i=0; i<BIN256; i++)
		{
			atomicAdd( &d_bin_data[i], d_bin_data_shared[i] );
		}
	}
#else
	// Chj: This is much better, all threads in current block are utilized.
	// On my RTX 3050 card, with GeForce driver 536.40, 
	// running `myhistogram 1024000 512`, myhistogram_03a is 20X the speed of myhistogram_02.
	//
	// But, on my GTX 870M, with GeForce driver 376.54, 
	// running `myhistogram 1024000 512`, myhistogram_03a is merely 20% faster than myhistogram_02.
	threads_per_block = blockDim.x * blockDim.y;
	idxBin = threadIdx.x;
	while( idxBin < BIN256 )
	{
		atomicAdd( &d_bin_data[idxBin], d_bin_data_shared[idxBin]);
		idxBin += threads_per_block;
	}
#endif
}

//////////////////////////////////////////////////////////////////////

void generate_histogram(const char *title, int sample_count, int threads_per_block)
{
	int i;
	Uchar *caSamples = new Uchar[sample_count]; // cpu mem
	Uchar *kaSamples = nullptr; // gpu mem
	Uint caCount_init[BIN256] = {}; // histogram init counted, the correct answer
	Uint caCount[BIN256] = {}; 
	Uint *kaCount = nullptr;      // histogram counted by gpu
	
	hipEvent_t start = nullptr, stop = nullptr; // for GPU timing
	HANDLE_ERROR( hipEventCreate(&start) );
	HANDLE_ERROR( hipEventCreate(&stop) );

	// fill caSamples[] and caCount_init[]
	//
	for(i=0; i<sample_count; i++)
	{
		int ball = rand() % BIN256;
		caSamples[i] = ball;
		caCount_init[ball]++ ;
	}

	printf("[%s] Counting %d samples ...\n", title, sample_count);

	// Copy host-RAM to gpu-RAM

	HANDLE_ERROR( hipMalloc((void**)&kaSamples, sample_count) );
	HANDLE_ERROR( hipMemcpy(kaSamples, caSamples, sample_count, hipMemcpyHostToDevice) );

	HANDLE_ERROR( hipMalloc((void**)&kaCount, BIN256*sizeof(int)) );
	HANDLE_ERROR( hipMemcpy(kaCount, caCount, BIN256*sizeof(int), hipMemcpyHostToDevice) );

	// start kernel-call timing
	HANDLE_ERROR( hipEventRecord( start, 0 ) ); 

	//
	// Select a kernel function to execute, according to `title`
	//

	if(strcmp(title, "p98:myhistogram_01")==0)
	{
		myhistogram_01<<<OCC_DIVIDE(sample_count, threads_per_block), threads_per_block>>>
			(kaSamples, kaCount, sample_count);
	}
	else if(strcmp(title, "p99:myhistogram_02")==0)
	{
		if(sample_count%4 != 0)
		{
			printf("ERROR user parameter input: For %s, sample_count must be multiple of 4. You passed in %d.\n",
				title, sample_count);
			exit(1);
		}

		int sample_ints = sample_count/4;
		myhistogram_02<<<OCC_DIVIDE(sample_ints, threads_per_block), threads_per_block>>>
			((Uint*)kaSamples, kaCount, sample_ints);
	}
	else if(strcmp(title, "p101:myhistogram_03a")==0)
	{
		if(sample_count%4 != 0)
		{
			printf("ERROR user parameter input: For %s, sample_count must be multiple of 4. You passed in %d.\n",
				title, sample_count);
			exit(1);
		}

		int sample_ints = sample_count/4;
		myhistogram_03a<<<OCC_DIVIDE(sample_ints, threads_per_block), threads_per_block>>>
			((Uint*)kaSamples, kaCount, sample_ints);
	}
	else
	{
		printf("ERROR: Unknown title requested: %s\n", title);
		exit(1);
	}

	// Check kernel launch success/fail.

	hipError_t kerr = hipPeekAtLastError();
	if(kerr) {
		printf("[%s] ERROR launching kernel call, errcode: %d (%s)\n", title, 
			kerr, hipGetErrorString(kerr));
		exit(4);
	}

	// stop kernel-call timing
	HANDLE_ERROR( hipEventRecord( stop, 0 ) ); 
	HANDLE_ERROR( hipEventSynchronize( stop ) );

	// Copy gpu-RAM to host-RAM (acquire result)

	HANDLE_ERROR( hipMemcpy(caCount, kaCount, BIN256*sizeof(int), hipMemcpyDeviceToHost) );

	const char *errprefix = nullptr;

	// Verify GPU-counted result.
	//
	printf("Verifying... ");
	for(i=0; i<BIN256; i++)
	{
		if(caCount[i]!=caCount_init[i])
		{
			printf("ERROR at sample index %d, correct: %d , wrong: %d\n",
				i, caCount_init[i], caCount[i]);
			
			errprefix = "Error!!!";
			break;
		}
	}

	float elapse_millisec = 0;
	HANDLE_ERROR( hipEventElapsedTime( &elapse_millisec, start, stop ) );

	if(elapse_millisec==0)
	{
		printf("%s (0 millisec)\n", 
			errprefix ? errprefix : "Success.");
	}
	else
	{
		printf("%s (%.5g millisec, %.5g GB/s)\n", 
			errprefix ? errprefix : "Success.",
			elapse_millisec, 
			((double)sample_count/(1000*1000))/elapse_millisec);
	}

	// Release resources.
	HANDLE_ERROR( hipFree(kaCount) );
	HANDLE_ERROR( hipFree(kaSamples) );
	delete caSamples;
}


extern"C" void 
main_myhistogram(int argc, char* argv[])
{
	if(argc==1)
	{
		printf("Usage:\n");
		printf("    myhistogram <sample_count> [threads_per_block]\n");
		printf("\n");
		printf("Examples:\n");
		printf("    myhistogram 1024\n");
		printf("    myhistogram 1024000 512\n");
		exit(1);
	}

	int sample_count = strtoul(argv[1], nullptr, 0);

	int threads_per_block = 256;

	if(argc>2) {
		threads_per_block = strtoul(argv[2], nullptr, 0);
	}
	
	if(sample_count<=0) {
		printf("Wrong sample_count number(must >0): %d\n", sample_count);
		exit(1);
	}

	if(threads_per_block<=0) {
		printf("Wrong threads_per_block number(must >0): %d\n", threads_per_block);
	}

	generate_histogram("p98:myhistogram_01", sample_count, threads_per_block);
	printf("\n");
	generate_histogram("p99:myhistogram_02", sample_count, threads_per_block);
	printf("\n");
	generate_histogram("p101:myhistogram_03a", sample_count, threads_per_block);
}

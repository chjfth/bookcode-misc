#include "hip/hip_runtime.h"
/*  Orignial code from [CUDA2012] Shane Cook's book, CH05, page 97-103.
2023.09.27, Chj adds caller code to the kernel functions and applies many fixes 
to his original kernel code. Now it runs smoothly, and the program's behavior
quite appositely matches the author's words.
*/

#include <stdio.h>
#include <stdlib.h>

#include "../../share/share.h"
#include "mykernels.h"

__shared__ unsigned int d_bin_data_shared[BIN256];
// -- sharedmem used by #03a, #03b and #07


void ReportErrorIfNot4xSamples(const char *title, int sample_count)
{
	if(sample_count%4 != 0)
	{
		printf("ERROR user parameter input: For %s, sample_count must be multiple of 4. You passed in %d.\n",
			title, sample_count);
		exit(1);
	}
}

void generate_histogram_gpu(const char *title, int sample_count, int threads_per_block,
	int Nbatch)
{
	Uchar *caSamples = new Uchar[sample_count]; // cpu mem
	Uchar *kaSamples = nullptr; // gpu mem
	Uint caCount_init[BIN256] = {}; // histogram init counted, the correct answer
	Uint caCount[BIN256] = {}; 
	Uint *kaCount = nullptr;      // histogram counted by gpu
	
	hipEvent_t start = nullptr, stop = nullptr; // for GPU timing
	HANDLE_ERROR( hipEventCreate(&start) );
	HANDLE_ERROR( hipEventCreate(&stop) );

	// fill caSamples[] and caCount_init[]
	prepare_samples(caSamples, sample_count, caCount_init);

	printf("[%s] Counting %d samples ...\n", title, sample_count);

	// Copy host-RAM to gpu-RAM

	HANDLE_ERROR( hipMalloc((void**)&kaSamples, sample_count) );
	HANDLE_ERROR( hipMemcpy(kaSamples, caSamples, sample_count, hipMemcpyHostToDevice) );

	HANDLE_ERROR( hipMalloc((void**)&kaCount, BIN256*sizeof(int)) );
	HANDLE_ERROR( hipMemcpy(kaCount, caCount, BIN256*sizeof(int), hipMemcpyHostToDevice) );

	// start kernel-call timing
	HANDLE_ERROR( hipEventRecord( start, 0 ) ); 

	//
	// Select a kernel function to execute, according to `title`
	//

	if(strcmp(title, "p98:myhistogram_01")==0)
	{
		myhistogram_01<<<OCC_DIVIDE(sample_count, threads_per_block), threads_per_block>>>
			(kaSamples, kaCount, sample_count);
	}
	else if(strcmp(title, "p99:myhistogram_02")==0)
	{
		ReportErrorIfNot4xSamples(title, sample_count);

		int sample_ints = sample_count/4;
		myhistogram_02<<<OCC_DIVIDE(sample_ints, threads_per_block), threads_per_block>>>
			((Uint*)kaSamples, kaCount, sample_ints);
	}
	else if(strcmp(title, "p101:myhistogram_03a")==0)
	{
		ReportErrorIfNot4xSamples(title, sample_count);

		int sample_ints = sample_count/4;
		myhistogram_03a<<<OCC_DIVIDE(sample_ints, threads_per_block), threads_per_block>>>
			((Uint*)kaSamples, kaCount, sample_ints);
	}
	else if(strcmp(title, "myhistogram_03b")==0)
	{
		myhistogram_03b<<<OCC_DIVIDE(sample_count, threads_per_block), threads_per_block>>>
			(kaSamples, kaCount, sample_count);
	}
	else if(strcmp(title, "p102:myhistogram_07")==0)
	{
		ReportErrorIfNot4xSamples(title, sample_count);

		printf("Using Nbatch = %d\n", Nbatch);

		int sample_ints = sample_count/4;
		Uint granularity = threads_per_block * Nbatch;

		myhistogram_07<<<OCC_DIVIDE(sample_ints, granularity), threads_per_block>>>
			((Uint*)kaSamples, kaCount, sample_ints, Nbatch);
	}
	else
	{
		printf("ERROR: Unknown GPU title requested: %s\n", title);
		exit(1);
	}

	// Check kernel launch success/fail.

	hipError_t kerr = hipPeekAtLastError();
	if(kerr) {
		printf("[%s] ERROR launching kernel call, errcode: %d (%s)\n", title, 
			kerr, hipGetErrorString(kerr));
		exit(4);
	}

	// stop kernel-call timing
	HANDLE_ERROR( hipEventRecord( stop, 0 ) ); 
	HANDLE_ERROR( hipEventSynchronize( stop ) );

	// Copy gpu-RAM to host-RAM (acquire result)

	HANDLE_ERROR( hipMemcpy(caCount, kaCount, BIN256*sizeof(int), hipMemcpyDeviceToHost) );

	const char *errprefix = nullptr;

	// Verify GPU-counted result.
	//
	printf("Verifying... ");
	bool vsucc = verify_bin_result(caCount_init, caCount);
	if(!vsucc)
		errprefix = "Error!!!";

	float elapse_millisec = 0;
	HANDLE_ERROR( hipEventElapsedTime( &elapse_millisec, start, stop ) );

	if(elapse_millisec==0)
	{
		printf("%s (0 millisec)\n", 
			errprefix ? errprefix : "Success.");
	}
	else
	{
		printf("%s (%.5g millisec, %.5g GB/s)\n", 
			errprefix ? errprefix : "Success.",
			elapse_millisec, 
			((double)sample_count/(1000*1000))/elapse_millisec);
	}

	// Release resources.
	HANDLE_ERROR( hipFree(kaCount) );
	HANDLE_ERROR( hipFree(kaSamples) );
	delete caSamples;
}


extern"C" void 
main_myhistogram(int argc, char* argv[])
{
	if(argc==1)
	{
		printf("Usage:\n");
		printf("    myhistogram <histogram_sample_count> [threads_per_block] [Nbatch]\n");
		printf("\n");
		printf("Examples:\n");
		printf("    myhistogram 1024\n");
		printf("    myhistogram 1024000 512\n");
		printf("    myhistogram 8 1 2\n");
		printf("    myhistogram 10240000 512 8\n");
		exit(1);
	}

	int sample_count = strtoul(argv[1], nullptr, 0);
	int threads_per_block = 256;
	int Nbatch = 1;

	if(argc>2) {
		threads_per_block = strtoul(argv[2], nullptr, 0);
	}
	if(argc>3) {
		// This param is used only 
		Nbatch = strtoul(argv[3], nullptr, 0);
	}
	
	if(sample_count<=0) {
		printf("Wrong sample_count number(must >0): %d\n", sample_count);
		exit(1);
	}
	if(threads_per_block<=0) {
		printf("Wrong threads_per_block number(must >0): %d\n", threads_per_block);
		exit(1);
	}
	if(Nbatch<=0) {
		printf("Wrong Nbatch number(must >0): %d\n", threads_per_block);
		exit(1);
	}

	generate_histogram_cpu("CPU_one_thread", sample_count);
	printf("\n");
	generate_histogram_cpu("CPU_two_threads", sample_count);
	printf("\n");

	myPrintGpuInfo();

	generate_histogram_gpu("p98:myhistogram_01", sample_count, threads_per_block, Nbatch);
	printf("\n");
	generate_histogram_gpu("p99:myhistogram_02", sample_count, threads_per_block, Nbatch);
	printf("\n");
	generate_histogram_gpu("p101:myhistogram_03a", sample_count, threads_per_block, Nbatch);
	printf("\n");
	generate_histogram_gpu("myhistogram_03b", sample_count, threads_per_block, Nbatch);
	printf("\n");
	generate_histogram_gpu("p102:myhistogram_07", sample_count, threads_per_block, Nbatch);
}

#include "hip/hip_runtime.h"
#include "mykernels.h"


__global__ void myhistogram_07( // @page 102
	const Uint * d_hist_data,
	Uint * d_bin_data,
	int sample_ints,
	Uint Nbatch) 
{
	// Chj: Note: this program still implies threadIdx.y==1

	/* Work out our thread id */
	Uint idx = (blockIdx.x * (blockDim.x*Nbatch)) + threadIdx.x;
	Uint idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	Uint tid = idx + idy * (blockDim.x*Nbatch) * gridDim.x;

	Uint idx_gpu = (blockIdx.x * blockDim.x) + threadIdx.x;
	Uint idy_gpu = (blockIdx.y * blockDim.y) + threadIdx.y;
	Uint tid_gpu = idx + idy * blockDim.x * gridDim.x;
	Uint tidQ = tid_gpu * Nbatch;
	
	Uint threads_per_block = blockDim.x * blockDim.y;

	if(tid!=tidQ)
	{
		printf("[b=%d,t=%d]Got bad.....(%d vs %d)\n", blockIdx.x, threadIdx.x,  tid, tidQ);
	}

	// Chj: Clear the d_bin_data_shared[] array.
	int idxBin = threadIdx.x;
	while(idxBin < BIN256)
	{
		d_bin_data_shared[idxBin] = 0;
		idxBin += threads_per_block;
	}

	// All threads should wait for the above clearing done.
	__syncthreads();

	// Fetch multiple histogram elements, and update poll-bin accordingly.
	//
	for(Uint i=0, tid_offset=0; 
		i<Nbatch; 
		i++, tid_offset += threads_per_block)
	{
		// note: tid_offset is counted in sizeof(Uint).

		Uint uint_offset = tidQ + tid_offset;

		if(uint_offset < sample_ints)
		{
			// Fetch four histogram elements in a group.
			Uint value_u32 = d_hist_data[uint_offset];

			atomicAdd( &(d_bin_data_shared[ (value_u32 & 0x000000FF) ]), 1 );
			atomicAdd( &(d_bin_data_shared[ (value_u32 & 0x0000FF00) >>  8 ]), 1 );
			atomicAdd( &(d_bin_data_shared[ (value_u32 & 0x00FF0000) >> 16 ]), 1 );
			atomicAdd( &(d_bin_data_shared[ (value_u32 & 0xFF000000) >> 24 ]), 1 );
		}
	}

	/* Wait for all threads to update shared memory, again */
	__syncthreads();

	// Merge block result in d_bin_data_shared[] into d_bin_data[]
	idxBin = threadIdx.x;
	while( idxBin < BIN256 )
	{
		atomicAdd( &d_bin_data[idxBin], d_bin_data_shared[idxBin]);
		idxBin += threads_per_block;
	}
}
